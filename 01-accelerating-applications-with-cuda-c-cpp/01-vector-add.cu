
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  for(int i = 0; i < N; ++i)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  /*
  a = (float *)malloc(size);
  b = (float *)malloc(size);
  c = (float *)malloc(size);
  */

  checkCuda(hipMallocManaged(&a, size));
  checkCuda(hipMallocManaged(&b, size));
  checkCuda(hipMallocManaged(&c, size));

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  //addVectorsInto(c, a, b, N);
  
  addVectorsInto<<<1, 1>>>(c, a, b, N);
  
  checkCuda(hipGetLastError());
  
  checkCuda(hipDeviceSynchronize());

  checkElementsAre(7, c, N);

  /*
  free(a);
  free(b);
  free(c);
  */

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
