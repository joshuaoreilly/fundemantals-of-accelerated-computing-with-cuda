#include "hip/hip_runtime.h"
#include <stdio.h>

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

__global__
void initWith(float num, float *a, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  hipDeviceProp_t props;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&props, deviceId);
  int multiProcessorCount = props.multiProcessorCount;

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  
  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  /*
   * nsys should register performance changes when execution configuration
   * is updated.
   */

  threadsPerBlock = 1024;
  numberOfBlocks = multiProcessorCount; //(N + threadsPerBlock - 1) / threadsPerBlock;

  printf("Grid size: %zu, Block size: %zu\n", numberOfBlocks, threadsPerBlock);

  hipError_t addVectorsErr;
  hipError_t asyncErr;
  
  initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);
  
  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
  
  hipMemPrefetchAsync(c, size, hipCpuDeviceId);

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
