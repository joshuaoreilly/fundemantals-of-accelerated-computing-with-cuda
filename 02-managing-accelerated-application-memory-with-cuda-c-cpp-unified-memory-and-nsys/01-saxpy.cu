#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define N 2048 * 2048 // Number of elements in each vector

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( tid < N )
        c[tid] = 2 * a[tid] + b[tid];
}

int main()
{
    int deviceId;
    hipDeviceProp_t props;
    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&props, deviceId);
    int multiProcessorCount = props.multiProcessorCount;

    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    checkCuda(hipMallocManaged(&a, size));
    checkCuda(hipMallocManaged(&b, size));
    checkCuda(hipMallocManaged(&c, size));

    // Initialize memory
    for( int i = 0; i < N; ++i )
    {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }

    int threads_per_block = 256;
    int number_of_blocks = (N / threads_per_block) + 1;

    printf("Grid size: %d, Block size: %d\n", number_of_blocks, threads_per_block);
    
    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    
    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
